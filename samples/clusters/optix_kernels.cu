#include "hip/hip_runtime.h"
﻿#include "clusters_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    uint32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryData geomData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_RG_NAME(raygen)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float>(launchIndex.x + plp.subPixelOffset.x) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + plp.subPixelOffset.y) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    uint32_t instIndex;
    uint32_t clusterId;
    uint32_t primIdx;
    float2 barycentrics;
    float3 shadingNormal;
    float3 geomNormal;
    MyPayloadSignature::trace(
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        instIndex, clusterId, primIdx, barycentrics, shadingNormal, geomNormal);

    if (launchIndex == plp.mousePosition) {
        plp.pickInfo->instanceIndex = instIndex;
        plp.pickInfo->clusterId = clusterId;
        plp.pickInfo->primitiveIndex = primIdx;
        plp.pickInfo->barycentrics = barycentrics;
        if (clusterId != OPTIX_CLUSTER_ID_INVALID) {
            plp.pickInfo->cluster = plp.clusters[clusterId];
        }
        else {
            plp.pickInfo->cluster.level = 0;
            plp.pickInfo->cluster.vertexCount = 0;
            plp.pickInfo->cluster.triangleCount = 0;
        }
    }

    bool hit = geomNormal != make_float3(0, 0, 0);
    float3 color = make_float3(0.0f, 0.0f, 0.1f);
    if (hit) {
        if (plp.visMode == VisualizationMode_ShadingNormal) {
            color = 0.5f * shadingNormal + make_float3(0.5f);
        }
        else if (plp.visMode == VisualizationMode_GeometricNormal) {
            color = 0.5f * geomNormal + make_float3(0.5f);
        }
        else if (plp.visMode == VisualizationMode_Cluster) {
            if (clusterId == OPTIX_CLUSTER_ID_INVALID) {
                color = make_float3(0.0f, 0.0f, 0.0f);
            }
            else {
                const float GoldenRatio = (1 + std::sqrt(5.0f)) / 2;
                const float GoldenAngle = 2 * pi_v<float> / (GoldenRatio * GoldenRatio);
                color = HSVtoRGB(
                    std::fmod((GoldenAngle * clusterId) / (2 * pi_v<float>), 1.0f),
                    1.0f, 1.0f);
            }
        }
        else if (plp.visMode == VisualizationMode_Level) {
            if (clusterId == OPTIX_CLUSTER_ID_INVALID) {
                color = make_float3(0.0f, 0.0f, 0.0f);
            }
            else {
                color = calcFalseColor(plp.clusters[clusterId].level, 0, 10);
            }
        }
    }

    float3 prevColorResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.sampleIndex > 0)
        prevColorResult = getXYZ(plp.colorAccumBuffer.read(launchIndex));
    float curWeight = 1.0f / (1 + plp.sampleIndex);
    float3 colorResult = (1 - curWeight) * prevColorResult + curWeight * color;
    plp.colorAccumBuffer.write(launchIndex, make_float4(colorResult, 1.0f));
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss)() {
    constexpr uint32_t instIndex = 0xFFFF'FFFF;
    constexpr uint32_t clusterId = OPTIX_CLUSTER_ID_INVALID;
    constexpr uint32_t primIdx = 0xFFFF'FFFF;
    constexpr float2 barycentrics = { 0.0f, 0.0f };
    float3 shadingNormal = make_float3(0, 0, 0);
    float3 geomNormal = make_float3(0, 0, 0);
    MyPayloadSignature::set(&instIndex, &clusterId, &primIdx, &barycentrics, &shadingNormal, &geomNormal);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(closesthit)() {
    const uint32_t clusterId = optixGetClusterId();
    const auto hp = HitPointParameter::get();

    const Cluster &cluster = plp.clusters[clusterId];
    const LocalTriangle &tri = plp.trianglePool[cluster.triPoolStartIndex + hp.primIndex];
    const Vertex (&vs)[] = {
        plp.vertexPool[cluster.vertPoolStartIndex + tri.index0],
        plp.vertexPool[cluster.vertPoolStartIndex + tri.index1],
        plp.vertexPool[cluster.vertPoolStartIndex + tri.index2],
    };

    float bcB = hp.b1;
    float bcC = hp.b2;
    float bcA = 1.0f - bcB - bcC;
    float3 shadingNormal =
        bcA * vs[0].normal + bcB * vs[1].normal + bcC * vs[2].normal;
    shadingNormal = normalize(optixTransformNormalFromObjectToWorldSpace(shadingNormal));

    //float3 positions[3];
    //optixGetTriangleVertexData(positions);
    //float3 geomNormal = normalize(cross(
    //    positions[1] - positions[0],
    //    positions[2] - positions[0]));
    float3 geomNormal = normalize(cross(
        vs[1].position - vs[0].position,
        vs[2].position - vs[0].position));
    geomNormal = normalize(optixTransformNormalFromObjectToWorldSpace(geomNormal));

    uint32_t instIndex = optixGetInstanceIndex();
    float2 barycentrics = make_float2(hp.b1, hp.b2);
    MyPayloadSignature::set(&instIndex, &clusterId, &hp.primIndex, &barycentrics, &shadingNormal, &geomNormal);
}
