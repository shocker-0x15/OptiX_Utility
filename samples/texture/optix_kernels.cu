#include "hip/hip_runtime.h"
﻿#include "texture_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

/*
JP: CH/AH/ISプログラムにてoptixGetSbtDataPointer()で取得できるポインターの位置に
    GeometryInstanceAccelerationStructureのsetUserData(), setChildUserData(),
    GeometryInstanceのsetUserData(), MaterialのsetUserData()
    で設定したデータが順番に並んでいる(各データの相対的な開始位置は指定したアラインメントに従う)。
    各データの開始位置は前方のデータのサイズによって変わるので、例えば同じGASに属していても
    GASの子ごとのデータサイズが異なればGeometryInstanceのデータの開始位置は異なる可能性があることに注意。
    このサンプルではGASとGASの子達にはユーザーデータは設定していない。
EN: Data set by each of
    GeometryInstanceAccelerationStructure's setUserData() and setChildUserData(),
    GeometryInstance's setUserData(), Material's setUserData()
    line up in the order (Each relative offset follows the specified alignment)
    at the position pointed by optixGetSbtDataPointer() called in CH/AH/IS programs.
    Note that the start position of each data changes depending on the sizes of forward data.
    Therefore for example, the start positions of GeometryInstance's data are different
    if data sizes of GAS children are different even if those belong to the same GAS.
    This sample does not set user data to GAS, GAS's child.
*/
struct HitGroupSBTRecordData {
    GeometryData geomData;
    MaterialData matData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_RG_NAME(raygen)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float>(launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + 0.5f) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    float3 color;
    MyPayloadSignature::trace(
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        color);

    plp.resultBuffer[launchIndex] = make_float4(color, 1.0f);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss)() {
    float3 color = make_float3(0, 0, 0.1f);
    MyPayloadSignature::set(&color);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(closesthit)() {
    auto sbtr = HitGroupSBTRecordData::get();
    const MaterialData &mat = sbtr.matData;
    const GeometryData &geom = sbtr.geomData;
    auto hp = HitPointParameter::get();

    const Triangle &triangle = geom.triangleBuffer[hp.primIndex];
    const Vertex &v0 = geom.vertexBuffer[triangle.index0];
    const Vertex &v1 = geom.vertexBuffer[triangle.index1];
    const Vertex &v2 = geom.vertexBuffer[triangle.index2];

    float b0 = 1 - (hp.b1 + hp.b2);
    float2 texCoord = b0 * v0.texCoord + hp.b1 * v1.texCoord + hp.b2 * v2.texCoord;

    // JP: テクスチャーのサンプリングは純粋なCUDAの組み込み関数を使う。
    // EN: Use a pure CUDA intrinsic function to sample a texture.
    float3 color;
    if (mat.texture)
        color = getXYZ(tex2DLod<float4>(mat.texture, texCoord.x, texCoord.y, 0.0f));
    else
        color = mat.albedo;
    MyPayloadSignature::set(&color);
}
