#include "hip/hip_runtime.h"
﻿#include "single_level_instancing_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

/*
JP: CH/AH/ISプログラムにてoptixGetSbtDataPointer()で取得できるポインターの位置に
    GeometryInstanceAccelerationStructureのsetUserData(), setChildUserData(),
    GeometryInstanceのsetUserData(), MaterialのsetUserData()
    で設定したデータが順番に並んでいる(各データの相対的な開始位置は指定したアラインメントに従う)。
    各データの開始位置は前方のデータのサイズによって変わるので、例えば同じGASに属していても
    GASの子ごとのデータサイズが異なればGeometryInstanceのデータの開始位置は異なる可能性があることに注意。
    このサンプルではGASとGASの子達、Materialにはユーザーデータは設定していない。
EN: Data set by each of
    GeometryInstanceAccelerationStructure's setUserData() and setChildUserData(),
    GeometryInstance's setUserData(), Material's setUserData()
    line up in the order (Each relative offset follows the specified alignment)
    at the position pointed by optixGetSbtDataPointer() called in CH/AH/IS programs.
    Note that the start position of each data changes depending on the sizes of forward data.
    Therefore for example, the start positions of GeometryInstance's data are different
    if data sizes of GAS children are different even if those belong to the same GAS.
    This sample does not set user data to GAS, GAS's child and Material.
*/
struct HitGroupSBTRecordData {
    GeometryData geomData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_RG_NAME(raygen)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float>(launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + 0.5f) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    float3 color;
    MyPayloadSignature::trace(
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        color);

    plp.resultBuffer[launchIndex] = make_float4(color, 1.0f);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss)() {
    float3 color = make_float3(0, 0, 0.1f);
    MyPayloadSignature::set(&color);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(closesthit)() {
    auto sbtr = HitGroupSBTRecordData::get();
    const GeometryData &geom = sbtr.geomData;
    auto hp = HitPointParameter::get();

    const Triangle &triangle = geom.triangleBuffer[hp.primIndex];
    const Vertex &v0 = geom.vertexBuffer[triangle.index0];
    const Vertex &v1 = geom.vertexBuffer[triangle.index1];
    const Vertex &v2 = geom.vertexBuffer[triangle.index2];

    float b0 = 1 - (hp.b1 + hp.b2);
    float3 sn = b0 * v0.normal + hp.b1 * v1.normal + hp.b2 * v2.normal;

    /*
    JP: GeometryInstanceからGAS空間への変換とは違って、GAS空間からインスタンス空間
        (1段階インスタンシングの場合はワールド空間に相当)への変換は組み込み関数が用意されている。
    EN: There is an intrinsic function to transform from GAS space to instance space
        (corresponds to world space in single-level instancing case)
        unlike the transform from GeometryInstance to GAS space.
    */
    sn = normalize(optixTransformNormalFromObjectToWorldSpace(sn));

    // JP: 法線の可視化。
    // EN: Display normal visualization.
    float3 color = 0.5f * sn + make_float3(0.5f);
    MyPayloadSignature::set(&color);
}
