#include "hip/hip_runtime.h"
﻿#include "single_gas_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

/*
JP: CH/AH/ISプログラムにてoptixGetSbtDataPointer()で取得できるポインターの位置に
    GeometryInstanceAccelerationStructureのsetUserData(), setChildUserData(),
    GeometryInstanceのsetUserData(), MaterialのsetUserData()
    で設定したデータが順番に並んでいる(各データの相対的な開始位置は指定したアラインメントに従う)。
    各データの開始位置は前方のデータのサイズによって変わるので、例えば同じGASに属していても
    GASの子ごとのデータサイズが異なればGeometryInstanceのデータの開始位置は異なる可能性があることに注意。
    このサンプルではGASとGASの子達、Materialにはユーザーデータは設定していない。
EN: Data set by each of
    GeometryInstanceAccelerationStructure's setUserData() and setChildUserData(),
    GeometryInstance's setUserData(), Material's setUserData()
    line up in the order (Each relative offset follows the specified alignment)
    at the position pointed by optixGetSbtDataPointer() called in CH/AH/IS programs.
    Note that the start position of each data changes depending on the sizes of forward data.
    Therefore for example, the start positions of GeometryInstance's data are different
    if data sizes of GAS children are different even if those belong to the same GAS.
    This sample does not set user data to GAS, GAS's child and Material.
*/
struct HitGroupSBTRecordData {
    GeometryData geomData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_RG_NAME(raygen0)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float>(launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + 0.5f) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    /*
    * JP: ペイロードとともにトレースを呼び出す。
    *     レイトレースは各ペイロードの型の組み合わせ、つまりシグネチャーを定義する
    *     optixu::PayloadSignatureテンプレート型を通じて行う。
    *     ペイロード数は最大で合計32DW。
    * EN: Trace call with payloads.
    *     Ray trace is done through the optixu::PayloadSignature template type
    *     which defines a combination of payload types, that is a signature.
    *     The maximum number of payloads is 32 dwords in total.
    */
    float3 color;
    MyPayloadSignature::trace(
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        color);

    plp.resultBuffer[launchIndex] = make_float4(color, 1.0f);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss0)() {
    float3 color = make_float3(0, 0, 0.1f);

    /*
    JP: PayloadSignature<...>::trace() と同じペイロードシグネチャー型を通じて返すペイロードをセットする。
        書き換えていないペイロードに関してはnullポインターを渡しても良い。
    EN: Set the returning payloads via the same payload signature type as PayloadSignature<...>::trace().
        Passing the null pointers is possible for the payloads which were read only.
    */
    MyPayloadSignature::set(&color);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(closesthit0)() {
    auto sbtr = HitGroupSBTRecordData::get();
    const GeometryData &geom = sbtr.geomData;
    auto hp = HitPointParameter::get();

    Triangle triangle;
    if (geom.triangleBuffer)
        triangle = geom.triangleBuffer[hp.primIndex];
    else // triangle soup
        triangle = Triangle{ 3u * hp.primIndex + 0, 3u * hp.primIndex + 1, 3u * hp.primIndex + 2 };
    const Vertex &v0 = geom.vertexBuffer[triangle.index0];
    const Vertex &v1 = geom.vertexBuffer[triangle.index1];
    const Vertex &v2 = geom.vertexBuffer[triangle.index2];

    float b0 = 1 - (hp.b1 + hp.b2);
    float3 sn = b0 * v0.normal + hp.b1 * v1.normal + hp.b2 * v2.normal;

    /*
    JP: GeometryInstanceからGAS空間への変換は自前で実装する必要がある。
    EN: Transform from GeometryInstance to GAS space should be manually implemented by the user.
    */
    sn = normalize(geom.transformNormal(sn));

    /*
    JP: 法線を可視化。
        このサンプルでは単一のGASしか使っていないためオブジェクト空間からワールド空間への変換は無い。
    EN: Visualize the normal.
        There is no object to world space transform since this sample uses only a single GAS.
    */
    float3 color = 0.5f * sn + make_float3(0.5f);

    /*
    JP: PayloadSignature<...>::trace() と同じペイロードシグネチャー型を通じて返すペイロードをセットする。
        書き換えていないペイロードに関してはnullポインターを渡しても良い。
    EN: Set the returning payloads via the same payload signature type as PayloadSignature<...>::trace().
        Passing the null pointers is possible for the payloads which were read only.
    */
    MyPayloadSignature::set(&color);
}
