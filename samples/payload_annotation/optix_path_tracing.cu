#include "hip/hip_runtime.h"
﻿#pragma once

#include "payload_annotation_shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;



struct HitPointParameter {
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get() {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData {
    GeometryData geomData;
    MaterialData matData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData &get() {
        return *reinterpret_cast<HitGroupSBTRecordData*>(optixGetSbtDataPointer());
    }
};



CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTracing)() {
    uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    PCG32RNG rng = plp.rngBuffer.read(launchIndex);

    float x = static_cast<float>(launchIndex.x + rng.getFloat0cTo1o()) / plp.imageSize.x;
    float y = static_cast<float>(launchIndex.y + rng.getFloat0cTo1o()) / plp.imageSize.y;
    float vh = 2 * std::tan(plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize(plp.camera.orientation * make_float3(vw * (0.5f - x), vh * (0.5f - y), 1));

    float3 accContribution = make_float3(0.0f, 0.0f, 0.0f);
    float3 accAlpha = make_float3(1.0f, 1.0f, 1.0f);
    PathFlags flags;
    flags.pathLength = 1;
    flags.terminate = false;
    while (true) {
        float3 contribution;
        float3 alpha;
        // JP: 通常のレイ用のペイロードタイプを指定する。
        // EN: Specify a payload type for the normal ray.
        constexpr OptixPayloadTypeID payloadTypeID = usePayloadAnnotation ?
            OPTIX_PAYLOAD_TYPE_ID_0 :
            OPTIX_PAYLOAD_TYPE_DEFAULT;
        SearchRayPayloadSignature::trace<payloadTypeID>(
            plp.travHandle, origin, direction,
            0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Search, NumRayTypes, RayType_Search,
            rng, alpha, contribution, origin, direction, flags);
        accContribution += accAlpha * contribution;
        accAlpha *= alpha;
        if (flags.terminate || flags.pathLength >= 10)
            break;
        ++flags.pathLength;
    }

    plp.rngBuffer.write(launchIndex, rng);

    float3 prevColorResult = make_float3(0.0f, 0.0f, 0.0f);
    if (plp.numAccumFrames > 0)
        prevColorResult = getXYZ(plp.colorAccumBuffer.read(launchIndex));
    float curWeight = 1.0f / (1 + plp.numAccumFrames);
    float3 colorResult = (1 - curWeight) * prevColorResult + curWeight * accContribution;
    plp.colorAccumBuffer.write(launchIndex, make_float4(colorResult, 1.0f));
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(miss)() {
    // JP: 通常のレイ用のペイロードタイプを指定する。
    // EN: Specify a payload type for the normal ray.
    if constexpr (usePayloadAnnotation)
        optixSetPayloadTypes(OPTIX_PAYLOAD_TYPE_ID_0);

    PathFlags flags;
    //SearchRayPayloadSignature::get(nullptr, nullptr, nullptr, nullptr, nullptr, &flags);
    SearchRayPayloadSignature::getAt<5>(&flags);
    float3 contribution = make_float3(0.01f, 0.01f, 0.01f);
    flags.terminate = true;
    //SearchRayPayloadSignature::set(nullptr, nullptr, &contribution, nullptr, nullptr, &flags);
    SearchRayPayloadSignature::setAt<2>(contribution);
    SearchRayPayloadSignature::setAt<5>(flags);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(shading)() {
    // JP: 通常のレイ用のペイロードタイプを指定する。
    // EN: Specify a payload type for the normal ray.
    if constexpr (usePayloadAnnotation)
        optixSetPayloadTypes(OPTIX_PAYLOAD_TYPE_ID_0);

    auto sbtr = HitGroupSBTRecordData::get();
    const MaterialData &mat = sbtr.matData;
    const GeometryData &geom = sbtr.geomData;

    PCG32RNG rng;
    PathFlags flags;
    //SearchRayPayloadSignature::get(&rng, nullptr, nullptr, nullptr, nullptr, &flags);
    SearchRayPayloadSignature::getAt<0>(&rng);
    SearchRayPayloadSignature::getAt<5>(&flags);

    auto hp = HitPointParameter::get();
    float3 p;
    float3 sn;
    float2 texCoord;
    {
        const Triangle &tri = geom.triangleBuffer[hp.primIndex];
        const Vertex &v0 = geom.vertexBuffer[tri.index0];
        const Vertex &v1 = geom.vertexBuffer[tri.index1];
        const Vertex &v2 = geom.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        p = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        sn = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        p = optixTransformPointFromObjectToWorldSpace(p);
        sn = normalize(optixTransformNormalFromObjectToWorldSpace(sn));
    }

    float3 vOut = -optixGetWorldRayDirection();
    bool isFrontFace = dot(vOut, sn) > 0;
    if (!isFrontFace)
        sn = -sn;
    p = p + sn * 0.001f;

    float3 albedo;
    if (mat.texture)
        albedo = getXYZ(tex2DLod<float4>(mat.texture, texCoord.x, texCoord.y, 0.0f));
    else
        albedo = mat.albedo;

    float3 contribution = make_float3(0.0f, 0.0f, 0.0f);

    const float3 LightRadiance = make_float3(30, 30, 30);
    // Hard-coded directly visible light
    if (mat.isEmitter && isFrontFace && flags.pathLength == 1)
        contribution += LightRadiance;

    // Next Event Estimation
    {
        // Use hard-coded area light for simplicity.
        float3 lp = make_float3(-0.25f, 0.9f, -0.25f) +
            rng.getFloat0cTo1o() * make_float3(0.5f, 0, 0) +
            rng.getFloat0cTo1o() * make_float3(0, 0, 0.5f);
        float areaPDF = 4.0f;
        float3 lpn = make_float3(0, -1, 0);

        float3 shadowRayDir = lp - p;
        float dist2 = dot(shadowRayDir, shadowRayDir);
        float dist = std::sqrt(dist2);
        shadowRayDir = shadowRayDir / dist;
        float cosLight = dot(lpn, -shadowRayDir);
        float3 Le = cosLight > 0 ? LightRadiance : make_float3(0, 0, 0);

        float visibility = 1.0f;
        // JP: シャドウレイ用のペイロードタイプを指定する。
        // EN: Specify a payload type for shadow ray.
        constexpr OptixPayloadTypeID payloadTypeID = usePayloadAnnotation ?
            OPTIX_PAYLOAD_TYPE_ID_1 :
            OPTIX_PAYLOAD_TYPE_DEFAULT;
        VisibilityRayPayloadSignature::trace<payloadTypeID>(
            plp.travHandle, p, shadowRayDir,
            0.0f, dist * 0.999f, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Visibility, NumRayTypes, RayType_Visibility,
            visibility);

        float cosSP = dot(sn, shadowRayDir);
        float G = visibility * std::fabs(cosSP) * std::fabs(cosLight) / dist2;
        float3 fs = cosSP > 0 ? albedo / Pi : make_float3(0, 0, 0);
        contribution += fs * G * Le / areaPDF;
    }

    const auto makeCoordinateSystem = [](const float3 &n, float3* s, float3* t) {
        float sign = n.z >= 0 ? 1 : -1;
        float a = -1 / (sign + n.z);
        float b = n.x * n.y * a;
        *s = make_float3(1 + sign * n.x * n.x * a, sign * b, -sign * n.x);
        *t = make_float3(b, sign + n.y * n.y * a, -n.y);
    };

    float3 s;
    float3 t;
    makeCoordinateSystem(sn, &s, &t);

    // Sampling incoming direction.
    float phi = 2 * Pi * rng.getFloat0cTo1o();
    float theta = std::asin(std::sqrt(rng.getFloat0cTo1o()));
    float sinTheta = std::sin(theta);
    float3 vIn = make_float3(std::cos(phi) * sinTheta, std::sin(phi) * sinTheta, std::cos(theta));
    vIn = make_float3(dot(make_float3(s.x, t.x, sn.x), vIn),
                      dot(make_float3(s.y, t.y, sn.y), vIn),
                      dot(make_float3(s.z, t.z, sn.z), vIn));
    float3 alpha = albedo;
    flags.terminate = false;

    SearchRayPayloadSignature::set(&rng, &alpha, &contribution, &p, &vIn, &flags);
}

CUDA_DEVICE_KERNEL void RT_AH_NAME(visibility)() {
    // JP: シャドウレイ用のペイロードタイプを指定する。
    // EN: Specify a payload type for shadow ray.
    if constexpr (usePayloadAnnotation)
        optixSetPayloadTypes(OPTIX_PAYLOAD_TYPE_ID_1);

    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set(&visibility);

    optixTerminateRay();
}
